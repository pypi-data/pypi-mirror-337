#include "hip/hip_runtime.h"
// See: ../deform2d_multiscale.cpp

#include <ATen/ATen.h>
#include <ATen/OpMathType.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/library.h>

#include <THC/THCAtomics.cuh>
#include <algorithm>
#include <cstdio>
#include <cstring>
#include <vector>

#include "utils.cuh"

namespace deform2d_multiscale {

const int64_t CUDA_NUM_THREADS = 1024;
inline int64_t GET_BLOCKS(const int64_t N, const int64_t num_threads) {
  return (N + num_threads - 1) / num_threads;
}

template <typename scalar_t>
__device__ scalar_t im2col_bilinear(const scalar_t *&bottom_data,
                                    const int64_t &height, const int64_t &width,
                                    const int64_t &nheads,
                                    const int64_t &channels, const scalar_t &h,
                                    const scalar_t &w, const int64_t &m,
                                    const int64_t &c) {
  const int64_t h_low = floor(h);
  const int64_t w_low = floor(w);
  const int64_t h_high = h_low + 1;
  const int64_t w_high = w_low + 1;

  const scalar_t lh = h - h_low;
  const scalar_t lw = w - w_low;
  const scalar_t hh = 1 - lh;
  const scalar_t hw = 1 - lw;

  const int64_t w_stride = nheads * channels;
  const int64_t h_stride = width * w_stride;
  const int64_t h_low_ptr_offset = h_low * h_stride;
  const int64_t h_high_ptr_offset = h_low_ptr_offset + h_stride;
  const int64_t w_low_ptr_offset = w_low * w_stride;
  const int64_t w_high_ptr_offset = w_low_ptr_offset + w_stride;
  const int64_t base_ptr = m * channels + c;

  scalar_t v1 = 0;
  if (h_low >= 0 && w_low >= 0) {
    const int64_t ptr1 = h_low_ptr_offset + w_low_ptr_offset + base_ptr;
    v1 = bottom_data[ptr1];
  }
  scalar_t v2 = 0;
  if (h_low >= 0 && w_high <= width - 1) {
    const int64_t ptr2 = h_low_ptr_offset + w_high_ptr_offset + base_ptr;
    v2 = bottom_data[ptr2];
  }
  scalar_t v3 = 0;
  if (h_high <= height - 1 && w_low >= 0) {
    const int64_t ptr3 = h_high_ptr_offset + w_low_ptr_offset + base_ptr;
    v3 = bottom_data[ptr3];
  }
  scalar_t v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1) {
    const int64_t ptr4 = h_high_ptr_offset + w_high_ptr_offset + base_ptr;
    v4 = bottom_data[ptr4];
  }

  const scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

  const scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  return val;
}

template <typename scalar_t>
__device__ void col2im_bilinear(
    const scalar_t *&bottom_data, const int64_t &height, const int64_t &width,
    const int64_t &nheads, const int64_t &channels, const scalar_t &h,
    const scalar_t &w, const int64_t &m, const int64_t &c,
    const scalar_t &top_grad, const scalar_t &attn_weight,
    scalar_t *&grad_value, scalar_t *grad_sampling_loc,
    scalar_t *grad_attn_weight) {
  const int64_t h_low = floor(h);
  const int64_t w_low = floor(w);
  const int64_t h_high = h_low + 1;
  const int64_t w_high = w_low + 1;

  const scalar_t lh = h - h_low;
  const scalar_t lw = w - w_low;
  const scalar_t hh = 1 - lh, hw = 1 - lw;

  const int64_t w_stride = nheads * channels;
  const int64_t h_stride = width * w_stride;
  const int64_t h_low_ptr_offset = h_low * h_stride;
  const int64_t h_high_ptr_offset = h_low_ptr_offset + h_stride;
  const int64_t w_low_ptr_offset = w_low * w_stride;
  const int64_t w_high_ptr_offset = w_low_ptr_offset + w_stride;
  const int64_t base_ptr = m * channels + c;

  const scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;
  const scalar_t top_grad_value = top_grad * attn_weight;
  scalar_t grad_h_weight = 0, grad_w_weight = 0;

  scalar_t v1 = 0;
  if (h_low >= 0 && w_low >= 0) {
    const int64_t ptr1 = h_low_ptr_offset + w_low_ptr_offset + base_ptr;
    v1 = bottom_data[ptr1];
    grad_h_weight -= hw * v1;
    grad_w_weight -= hh * v1;
    atomicAdd(grad_value + ptr1, w1 * top_grad_value);
  }
  scalar_t v2 = 0;
  if (h_low >= 0 && w_high <= width - 1) {
    const int64_t ptr2 = h_low_ptr_offset + w_high_ptr_offset + base_ptr;
    v2 = bottom_data[ptr2];
    grad_h_weight -= lw * v2;
    grad_w_weight += hh * v2;
    atomicAdd(grad_value + ptr2, w2 * top_grad_value);
  }
  scalar_t v3 = 0;
  if (h_high <= height - 1 && w_low >= 0) {
    const int64_t ptr3 = h_high_ptr_offset + w_low_ptr_offset + base_ptr;
    v3 = bottom_data[ptr3];
    grad_h_weight += hw * v3;
    grad_w_weight -= lh * v3;
    atomicAdd(grad_value + ptr3, w3 * top_grad_value);
  }
  scalar_t v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1) {
    const int64_t ptr4 = h_high_ptr_offset + w_high_ptr_offset + base_ptr;
    v4 = bottom_data[ptr4];
    grad_h_weight += lw * v4;
    grad_w_weight += lh * v4;
    atomicAdd(grad_value + ptr4, w4 * top_grad_value);
  }

  const scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  *grad_attn_weight = top_grad * val;
  *grad_sampling_loc = width * grad_w_weight * top_grad_value;
  *(grad_sampling_loc + 1) = height * grad_h_weight * top_grad_value;
}

template <typename scalar_t>
__device__ void col2im_bilinear_gm(
    const scalar_t *&bottom_data, const int64_t &height, const int64_t &width,
    const int64_t &nheads, const int64_t &channels, const scalar_t &h,
    const scalar_t &w, const int64_t &m, const int64_t &c,
    const scalar_t &top_grad, const scalar_t &attn_weight,
    scalar_t *&grad_value, scalar_t *grad_sampling_loc,
    scalar_t *grad_attn_weight) {
  const int64_t h_low = floor(h);
  const int64_t w_low = floor(w);
  const int64_t h_high = h_low + 1;
  const int64_t w_high = w_low + 1;

  const scalar_t lh = h - h_low;
  const scalar_t lw = w - w_low;
  const scalar_t hh = 1 - lh, hw = 1 - lw;

  const int64_t w_stride = nheads * channels;
  const int64_t h_stride = width * w_stride;
  const int64_t h_low_ptr_offset = h_low * h_stride;
  const int64_t h_high_ptr_offset = h_low_ptr_offset + h_stride;
  const int64_t w_low_ptr_offset = w_low * w_stride;
  const int64_t w_high_ptr_offset = w_low_ptr_offset + w_stride;
  const int64_t base_ptr = m * channels + c;

  const scalar_t w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;
  const scalar_t top_grad_value = top_grad * attn_weight;
  scalar_t grad_h_weight = 0, grad_w_weight = 0;

  scalar_t v1 = 0;
  if (h_low >= 0 && w_low >= 0) {
    const int64_t ptr1 = h_low_ptr_offset + w_low_ptr_offset + base_ptr;
    v1 = bottom_data[ptr1];
    grad_h_weight -= hw * v1;
    grad_w_weight -= hh * v1;
    atomicAdd(grad_value + ptr1, w1 * top_grad_value);
  }
  scalar_t v2 = 0;
  if (h_low >= 0 && w_high <= width - 1) {
    const int64_t ptr2 = h_low_ptr_offset + w_high_ptr_offset + base_ptr;
    v2 = bottom_data[ptr2];
    grad_h_weight -= lw * v2;
    grad_w_weight += hh * v2;
    atomicAdd(grad_value + ptr2, w2 * top_grad_value);
  }
  scalar_t v3 = 0;
  if (h_high <= height - 1 && w_low >= 0) {
    const int64_t ptr3 = h_high_ptr_offset + w_low_ptr_offset + base_ptr;
    v3 = bottom_data[ptr3];
    grad_h_weight += hw * v3;
    grad_w_weight -= lh * v3;
    atomicAdd(grad_value + ptr3, w3 * top_grad_value);
  }
  scalar_t v4 = 0;
  if (h_high <= height - 1 && w_high <= width - 1) {
    const int64_t ptr4 = h_high_ptr_offset + w_high_ptr_offset + base_ptr;
    v4 = bottom_data[ptr4];
    grad_h_weight += lw * v4;
    grad_w_weight += lh * v4;
    atomicAdd(grad_value + ptr4, w4 * top_grad_value);
  }

  const scalar_t val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
  atomicAdd(grad_attn_weight, top_grad * val);
  atomicAdd(grad_sampling_loc, width * grad_w_weight * top_grad_value);
  atomicAdd(grad_sampling_loc + 1, height * grad_h_weight * top_grad_value);
}

template <typename scalar_t>
__global__ void deform_im2col_gpu_kernel(
    const int64_t n, const scalar_t *data_value,
    const int64_t *data_spatial_shapes, const int64_t *data_level_start_index,
    const scalar_t *data_sampling_loc, const scalar_t *data_attn_weight,
    const int64_t batch_size, const int64_t spatial_size,
    const int64_t num_heads, const int64_t channels, const int64_t num_levels,
    const int64_t num_query, const int64_t num_point, scalar_t *data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    int64_t _temp = index;
    const int64_t c_col = _temp % channels;
    _temp /= channels;
    const int64_t sampling_index = _temp;
    const int64_t m_col = _temp % num_heads;
    _temp /= num_heads;
    // const int64_t q_col = _temp % num_query;
    _temp /= num_query;
    const int64_t b_col = _temp;

    scalar_t *data_col_ptr = data_col + index;
    int64_t data_weight_ptr = sampling_index * num_levels * num_point;
    int64_t data_loc_w_ptr = data_weight_ptr << 1;
    const int64_t qid_stride = num_heads * channels;
    const int64_t data_value_ptr_init_offset =
        b_col * spatial_size * qid_stride;
    scalar_t col = 0;

    for (int l_col = 0; l_col < num_levels; ++l_col) {
      const int64_t level_start_id = data_level_start_index[l_col];
      const int64_t spatial_h_ptr = l_col << 1;
      const int64_t spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int64_t spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const scalar_t *data_value_ptr =
          data_value +
          (data_value_ptr_init_offset + level_start_id * qid_stride);
      for (int p_col = 0; p_col < num_point; ++p_col) {
        const scalar_t loc_w = data_sampling_loc[data_loc_w_ptr];
        const scalar_t loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const scalar_t weight = data_attn_weight[data_weight_ptr];

        const scalar_t h_im = loc_h * spatial_h - 0.5;
        const scalar_t w_im = loc_w * spatial_w - 0.5;

        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w) {
          col +=
              im2col_bilinear(data_value_ptr, spatial_h, spatial_w, num_heads,
                              channels, h_im, w_im, m_col, c_col) *
              weight;
        }

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
      }
    }
    *data_col_ptr = col;
  }
}

template <typename scalar_t, unsigned int blockSize>
__global__ void deform_col2im_gpu_kernel_shm_blocksize_aware_reduce_v1(
    const int64_t n, const scalar_t *grad_col, const scalar_t *data_value,
    const int64_t *data_spatial_shapes, const int64_t *data_level_start_index,
    const scalar_t *data_sampling_loc, const scalar_t *data_attn_weight,
    const int64_t batch_size, const int64_t spatial_size,
    const int64_t num_heads, const int64_t channels, const int64_t num_levels,
    const int64_t num_query, const int64_t num_point, scalar_t *grad_value,
    scalar_t *grad_sampling_loc, scalar_t *grad_attn_weight) {
  CUDA_KERNEL_LOOP(index, n) {
    __shared__ scalar_t cache_grad_sampling_loc[blockSize * 2];
    __shared__ scalar_t cache_grad_attn_weight[blockSize];
    unsigned int tid = threadIdx.x;
    int64_t _temp = index;
    const int64_t c_col = _temp % channels;
    _temp /= channels;
    const int64_t sampling_index = _temp;
    const int64_t m_col = _temp % num_heads;
    _temp /= num_heads;
    // const int64_t q_col = _temp % num_query;
    _temp /= num_query;
    const int64_t b_col = _temp;

    const scalar_t top_grad = grad_col[index];

    int64_t data_weight_ptr = sampling_index * num_levels * num_point;
    int64_t data_loc_w_ptr = data_weight_ptr << 1;
    const int64_t grad_sampling_ptr = data_weight_ptr;
    grad_sampling_loc += grad_sampling_ptr << 1;
    grad_attn_weight += grad_sampling_ptr;
    const int64_t grad_weight_stride = 1;
    const int64_t grad_loc_stride = 2;
    const int64_t qid_stride = num_heads * channels;
    const int64_t data_value_ptr_init_offset =
        b_col * spatial_size * qid_stride;

    for (int l_col = 0; l_col < num_levels; ++l_col) {
      const int64_t level_start_id = data_level_start_index[l_col];
      const int64_t spatial_h_ptr = l_col << 1;
      const int64_t spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int64_t spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const int64_t value_ptr_offset =
          data_value_ptr_init_offset + level_start_id * qid_stride;
      const scalar_t *data_value_ptr = data_value + value_ptr_offset;
      scalar_t *grad_value_ptr = grad_value + value_ptr_offset;

      for (int p_col = 0; p_col < num_point; ++p_col) {
        const scalar_t loc_w = data_sampling_loc[data_loc_w_ptr];
        const scalar_t loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const scalar_t weight = data_attn_weight[data_weight_ptr];

        const scalar_t h_im = loc_h * spatial_h - 0.5;
        const scalar_t w_im = loc_w * spatial_w - 0.5;
        *(cache_grad_sampling_loc + (threadIdx.x << 1)) = 0;
        *(cache_grad_sampling_loc + ((threadIdx.x << 1) + 1)) = 0;
        *(cache_grad_attn_weight + threadIdx.x) = 0;
        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w) {
          col2im_bilinear(data_value_ptr, spatial_h, spatial_w, num_heads,
                          channels, h_im, w_im, m_col, c_col, top_grad, weight,
                          grad_value_ptr,
                          cache_grad_sampling_loc + (threadIdx.x << 1),
                          cache_grad_attn_weight + threadIdx.x);
        }

        __syncthreads();
        if (tid == 0) {
          scalar_t _grad_w = cache_grad_sampling_loc[0],
                   _grad_h = cache_grad_sampling_loc[1],
                   _grad_a = cache_grad_attn_weight[0];
          int64_t sid = 2;
          for (unsigned int tid = 1; tid < blockSize; ++tid) {
            _grad_w += cache_grad_sampling_loc[sid];
            _grad_h += cache_grad_sampling_loc[sid + 1];
            _grad_a += cache_grad_attn_weight[tid];
            sid += 2;
          }

          *grad_sampling_loc = _grad_w;
          *(grad_sampling_loc + 1) = _grad_h;
          *grad_attn_weight = _grad_a;
        }
        __syncthreads();

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
        grad_attn_weight += grad_weight_stride;
        grad_sampling_loc += grad_loc_stride;
      }
    }
  }
}

template <typename scalar_t, unsigned int blockSize>
__global__ void deform_col2im_gpu_kernel_shm_blocksize_aware_reduce_v2(
    const int64_t n, const scalar_t *grad_col, const scalar_t *data_value,
    const int64_t *data_spatial_shapes, const int64_t *data_level_start_index,
    const scalar_t *data_sampling_loc, const scalar_t *data_attn_weight,
    const int64_t batch_size, const int64_t spatial_size,
    const int64_t num_heads, const int64_t channels, const int64_t num_levels,
    const int64_t num_query, const int64_t num_point, scalar_t *grad_value,
    scalar_t *grad_sampling_loc, scalar_t *grad_attn_weight) {
  CUDA_KERNEL_LOOP(index, n) {
    __shared__ scalar_t cache_grad_sampling_loc[blockSize * 2];
    __shared__ scalar_t cache_grad_attn_weight[blockSize];
    unsigned int tid = threadIdx.x;
    int64_t _temp = index;
    const int64_t c_col = _temp % channels;
    _temp /= channels;
    const int64_t sampling_index = _temp;
    const int64_t m_col = _temp % num_heads;
    _temp /= num_heads;
    // const int64_t q_col = _temp % num_query;
    _temp /= num_query;
    const int64_t b_col = _temp;

    const scalar_t top_grad = grad_col[index];

    int64_t data_weight_ptr = sampling_index * num_levels * num_point;
    int64_t data_loc_w_ptr = data_weight_ptr << 1;
    const int64_t grad_sampling_ptr = data_weight_ptr;
    grad_sampling_loc += grad_sampling_ptr << 1;
    grad_attn_weight += grad_sampling_ptr;
    const int64_t grad_weight_stride = 1;
    const int64_t grad_loc_stride = 2;
    const int64_t qid_stride = num_heads * channels;
    const int64_t data_value_ptr_init_offset =
        b_col * spatial_size * qid_stride;

    for (int l_col = 0; l_col < num_levels; ++l_col) {
      const int64_t level_start_id = data_level_start_index[l_col];
      const int64_t spatial_h_ptr = l_col << 1;
      const int64_t spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int64_t spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const int64_t value_ptr_offset =
          data_value_ptr_init_offset + level_start_id * qid_stride;
      const scalar_t *data_value_ptr = data_value + value_ptr_offset;
      scalar_t *grad_value_ptr = grad_value + value_ptr_offset;

      for (int p_col = 0; p_col < num_point; ++p_col) {
        const scalar_t loc_w = data_sampling_loc[data_loc_w_ptr];
        const scalar_t loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const scalar_t weight = data_attn_weight[data_weight_ptr];

        const scalar_t h_im = loc_h * spatial_h - 0.5;
        const scalar_t w_im = loc_w * spatial_w - 0.5;
        *(cache_grad_sampling_loc + (threadIdx.x << 1)) = 0;
        *(cache_grad_sampling_loc + ((threadIdx.x << 1) + 1)) = 0;
        *(cache_grad_attn_weight + threadIdx.x) = 0;
        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w) {
          col2im_bilinear(data_value_ptr, spatial_h, spatial_w, num_heads,
                          channels, h_im, w_im, m_col, c_col, top_grad, weight,
                          grad_value_ptr,
                          cache_grad_sampling_loc + (threadIdx.x << 1),
                          cache_grad_attn_weight + threadIdx.x);
        }

        __syncthreads();

        for (unsigned int s = blockSize / 2; s > 0; s >>= 1) {
          if (tid < s) {
            const unsigned int xid1 = tid << 1;
            const unsigned int xid2 = (tid + s) << 1;
            cache_grad_attn_weight[tid] += cache_grad_attn_weight[tid + s];
            cache_grad_sampling_loc[xid1] += cache_grad_sampling_loc[xid2];
            cache_grad_sampling_loc[xid1 + 1] +=
                cache_grad_sampling_loc[xid2 + 1];
          }
          __syncthreads();
        }

        if (tid == 0) {
          *grad_sampling_loc = cache_grad_sampling_loc[0];
          *(grad_sampling_loc + 1) = cache_grad_sampling_loc[1];
          *grad_attn_weight = cache_grad_attn_weight[0];
        }
        __syncthreads();

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
        grad_attn_weight += grad_weight_stride;
        grad_sampling_loc += grad_loc_stride;
      }
    }
  }
}

template <typename scalar_t>
__global__ void deform_col2im_gpu_kernel_shm_reduce_v1(
    const int64_t n, const scalar_t *grad_col, const scalar_t *data_value,
    const int64_t *data_spatial_shapes, const int64_t *data_level_start_index,
    const scalar_t *data_sampling_loc, const scalar_t *data_attn_weight,
    const int64_t batch_size, const int64_t spatial_size,
    const int64_t num_heads, const int64_t channels, const int64_t num_levels,
    const int64_t num_query, const int64_t num_point, scalar_t *grad_value,
    scalar_t *grad_sampling_loc, scalar_t *grad_attn_weight) {
  CUDA_KERNEL_LOOP(index, n) {
    extern __shared__ int64_t _s[];
    scalar_t *cache_grad_sampling_loc = (scalar_t *)_s;
    scalar_t *cache_grad_attn_weight = cache_grad_sampling_loc + 2 * blockDim.x;
    unsigned int tid = threadIdx.x;
    int64_t _temp = index;
    const int64_t c_col = _temp % channels;
    _temp /= channels;
    const int64_t sampling_index = _temp;
    const int64_t m_col = _temp % num_heads;
    _temp /= num_heads;
    // const int64_t q_col = _temp % num_query;
    _temp /= num_query;
    const int64_t b_col = _temp;

    const scalar_t top_grad = grad_col[index];

    int64_t data_weight_ptr = sampling_index * num_levels * num_point;
    int64_t data_loc_w_ptr = data_weight_ptr << 1;
    const int64_t grad_sampling_ptr = data_weight_ptr;
    grad_sampling_loc += grad_sampling_ptr << 1;
    grad_attn_weight += grad_sampling_ptr;
    const int64_t grad_weight_stride = 1;
    const int64_t grad_loc_stride = 2;
    const int64_t qid_stride = num_heads * channels;
    const int64_t data_value_ptr_init_offset =
        b_col * spatial_size * qid_stride;

    for (int l_col = 0; l_col < num_levels; ++l_col) {
      const int64_t level_start_id = data_level_start_index[l_col];
      const int64_t spatial_h_ptr = l_col << 1;
      const int64_t spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int64_t spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const int64_t value_ptr_offset =
          data_value_ptr_init_offset + level_start_id * qid_stride;
      const scalar_t *data_value_ptr = data_value + value_ptr_offset;
      scalar_t *grad_value_ptr = grad_value + value_ptr_offset;

      for (int p_col = 0; p_col < num_point; ++p_col) {
        const scalar_t loc_w = data_sampling_loc[data_loc_w_ptr];
        const scalar_t loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const scalar_t weight = data_attn_weight[data_weight_ptr];

        const scalar_t h_im = loc_h * spatial_h - 0.5;
        const scalar_t w_im = loc_w * spatial_w - 0.5;
        *(cache_grad_sampling_loc + (threadIdx.x << 1)) = 0;
        *(cache_grad_sampling_loc + ((threadIdx.x << 1) + 1)) = 0;
        *(cache_grad_attn_weight + threadIdx.x) = 0;
        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w) {
          col2im_bilinear(data_value_ptr, spatial_h, spatial_w, num_heads,
                          channels, h_im, w_im, m_col, c_col, top_grad, weight,
                          grad_value_ptr,
                          cache_grad_sampling_loc + (threadIdx.x << 1),
                          cache_grad_attn_weight + threadIdx.x);
        }

        __syncthreads();
        if (tid == 0) {
          scalar_t _grad_w = cache_grad_sampling_loc[0],
                   _grad_h = cache_grad_sampling_loc[1],
                   _grad_a = cache_grad_attn_weight[0];
          int64_t sid = 2;
          for (unsigned int tid = 1; tid < blockDim.x; ++tid) {
            _grad_w += cache_grad_sampling_loc[sid];
            _grad_h += cache_grad_sampling_loc[sid + 1];
            _grad_a += cache_grad_attn_weight[tid];
            sid += 2;
          }

          *grad_sampling_loc = _grad_w;
          *(grad_sampling_loc + 1) = _grad_h;
          *grad_attn_weight = _grad_a;
        }
        __syncthreads();

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
        grad_attn_weight += grad_weight_stride;
        grad_sampling_loc += grad_loc_stride;
      }
    }
  }
}

template <typename scalar_t>
__global__ void deform_col2im_gpu_kernel_shm_reduce_v2(
    const int64_t n, const scalar_t *grad_col, const scalar_t *data_value,
    const int64_t *data_spatial_shapes, const int64_t *data_level_start_index,
    const scalar_t *data_sampling_loc, const scalar_t *data_attn_weight,
    const int64_t batch_size, const int64_t spatial_size,
    const int64_t num_heads, const int64_t channels, const int64_t num_levels,
    const int64_t num_query, const int64_t num_point, scalar_t *grad_value,
    scalar_t *grad_sampling_loc, scalar_t *grad_attn_weight) {
  CUDA_KERNEL_LOOP(index, n) {
    extern __shared__ int64_t _s[];
    scalar_t *cache_grad_sampling_loc = (scalar_t *)_s;
    scalar_t *cache_grad_attn_weight = cache_grad_sampling_loc + 2 * blockDim.x;
    unsigned int tid = threadIdx.x;
    int64_t _temp = index;
    const int64_t c_col = _temp % channels;
    _temp /= channels;
    const int64_t sampling_index = _temp;
    const int64_t m_col = _temp % num_heads;
    _temp /= num_heads;
    // const int64_t q_col = _temp % num_query;
    _temp /= num_query;
    const int64_t b_col = _temp;

    const scalar_t top_grad = grad_col[index];

    int64_t data_weight_ptr = sampling_index * num_levels * num_point;
    int64_t data_loc_w_ptr = data_weight_ptr << 1;
    const int64_t grad_sampling_ptr = data_weight_ptr;
    grad_sampling_loc += grad_sampling_ptr << 1;
    grad_attn_weight += grad_sampling_ptr;
    const int64_t grad_weight_stride = 1;
    const int64_t grad_loc_stride = 2;
    const int64_t qid_stride = num_heads * channels;
    const int64_t data_value_ptr_init_offset =
        b_col * spatial_size * qid_stride;

    for (int l_col = 0; l_col < num_levels; ++l_col) {
      const int64_t level_start_id = data_level_start_index[l_col];
      const int64_t spatial_h_ptr = l_col << 1;
      const int64_t spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int64_t spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const int64_t value_ptr_offset =
          data_value_ptr_init_offset + level_start_id * qid_stride;
      const scalar_t *data_value_ptr = data_value + value_ptr_offset;
      scalar_t *grad_value_ptr = grad_value + value_ptr_offset;

      for (int p_col = 0; p_col < num_point; ++p_col) {
        const scalar_t loc_w = data_sampling_loc[data_loc_w_ptr];
        const scalar_t loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const scalar_t weight = data_attn_weight[data_weight_ptr];

        const scalar_t h_im = loc_h * spatial_h - 0.5;
        const scalar_t w_im = loc_w * spatial_w - 0.5;
        *(cache_grad_sampling_loc + (threadIdx.x << 1)) = 0;
        *(cache_grad_sampling_loc + ((threadIdx.x << 1) + 1)) = 0;
        *(cache_grad_attn_weight + threadIdx.x) = 0;
        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w) {
          col2im_bilinear(data_value_ptr, spatial_h, spatial_w, num_heads,
                          channels, h_im, w_im, m_col, c_col, top_grad, weight,
                          grad_value_ptr,
                          cache_grad_sampling_loc + (threadIdx.x << 1),
                          cache_grad_attn_weight + threadIdx.x);
        }

        __syncthreads();

        for (unsigned int s = blockDim.x / 2, spre = blockDim.x; s > 0;
             s >>= 1, spre >>= 1) {
          if (tid < s) {
            const unsigned int xid1 = tid << 1;
            const unsigned int xid2 = (tid + s) << 1;
            cache_grad_attn_weight[tid] += cache_grad_attn_weight[tid + s];
            cache_grad_sampling_loc[xid1] += cache_grad_sampling_loc[xid2];
            cache_grad_sampling_loc[xid1 + 1] +=
                cache_grad_sampling_loc[xid2 + 1];
            if (tid + (s << 1) < spre) {
              cache_grad_attn_weight[tid] +=
                  cache_grad_attn_weight[tid + (s << 1)];
              cache_grad_sampling_loc[xid1] +=
                  cache_grad_sampling_loc[xid2 + (s << 1)];
              cache_grad_sampling_loc[xid1 + 1] +=
                  cache_grad_sampling_loc[xid2 + 1 + (s << 1)];
            }
          }
          __syncthreads();
        }

        if (tid == 0) {
          *grad_sampling_loc = cache_grad_sampling_loc[0];
          *(grad_sampling_loc + 1) = cache_grad_sampling_loc[1];
          *grad_attn_weight = cache_grad_attn_weight[0];
        }
        __syncthreads();

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
        grad_attn_weight += grad_weight_stride;
        grad_sampling_loc += grad_loc_stride;
      }
    }
  }
}

template <typename scalar_t>
__global__ void deform_col2im_gpu_kernel_shm_reduce_v2_multi_blocks(
    const int64_t n, const scalar_t *grad_col, const scalar_t *data_value,
    const int64_t *data_spatial_shapes, const int64_t *data_level_start_index,
    const scalar_t *data_sampling_loc, const scalar_t *data_attn_weight,
    const int64_t batch_size, const int64_t spatial_size,
    const int64_t num_heads, const int64_t channels, const int64_t num_levels,
    const int64_t num_query, const int64_t num_point, scalar_t *grad_value,
    scalar_t *grad_sampling_loc, scalar_t *grad_attn_weight) {
  CUDA_KERNEL_LOOP(index, n) {
    extern __shared__ int64_t _s[];
    scalar_t *cache_grad_sampling_loc = (scalar_t *)_s;
    scalar_t *cache_grad_attn_weight = cache_grad_sampling_loc + 2 * blockDim.x;
    unsigned int tid = threadIdx.x;
    int64_t _temp = index;
    const int64_t c_col = _temp % channels;
    _temp /= channels;
    const int64_t sampling_index = _temp;
    const int64_t m_col = _temp % num_heads;
    _temp /= num_heads;
    // const int64_t q_col = _temp % num_query;
    _temp /= num_query;
    const int64_t b_col = _temp;

    const scalar_t top_grad = grad_col[index];

    int64_t data_weight_ptr = sampling_index * num_levels * num_point;
    int64_t data_loc_w_ptr = data_weight_ptr << 1;
    const int64_t grad_sampling_ptr = data_weight_ptr;
    grad_sampling_loc += grad_sampling_ptr << 1;
    grad_attn_weight += grad_sampling_ptr;
    const int64_t grad_weight_stride = 1;
    const int64_t grad_loc_stride = 2;
    const int64_t qid_stride = num_heads * channels;
    const int64_t data_value_ptr_init_offset =
        b_col * spatial_size * qid_stride;

    for (int l_col = 0; l_col < num_levels; ++l_col) {
      const int64_t level_start_id = data_level_start_index[l_col];
      const int64_t spatial_h_ptr = l_col << 1;
      const int64_t spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int64_t spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const int64_t value_ptr_offset =
          data_value_ptr_init_offset + level_start_id * qid_stride;
      const scalar_t *data_value_ptr = data_value + value_ptr_offset;
      scalar_t *grad_value_ptr = grad_value + value_ptr_offset;

      for (int p_col = 0; p_col < num_point; ++p_col) {
        const scalar_t loc_w = data_sampling_loc[data_loc_w_ptr];
        const scalar_t loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const scalar_t weight = data_attn_weight[data_weight_ptr];

        const scalar_t h_im = loc_h * spatial_h - 0.5;
        const scalar_t w_im = loc_w * spatial_w - 0.5;
        *(cache_grad_sampling_loc + (threadIdx.x << 1)) = 0;
        *(cache_grad_sampling_loc + ((threadIdx.x << 1) + 1)) = 0;
        *(cache_grad_attn_weight + threadIdx.x) = 0;
        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w) {
          col2im_bilinear(data_value_ptr, spatial_h, spatial_w, num_heads,
                          channels, h_im, w_im, m_col, c_col, top_grad, weight,
                          grad_value_ptr,
                          cache_grad_sampling_loc + (threadIdx.x << 1),
                          cache_grad_attn_weight + threadIdx.x);
        }

        __syncthreads();

        for (unsigned int s = blockDim.x / 2, spre = blockDim.x; s > 0;
             s >>= 1, spre >>= 1) {
          if (tid < s) {
            const unsigned int xid1 = tid << 1;
            const unsigned int xid2 = (tid + s) << 1;
            cache_grad_attn_weight[tid] += cache_grad_attn_weight[tid + s];
            cache_grad_sampling_loc[xid1] += cache_grad_sampling_loc[xid2];
            cache_grad_sampling_loc[xid1 + 1] +=
                cache_grad_sampling_loc[xid2 + 1];
            if (tid + (s << 1) < spre) {
              cache_grad_attn_weight[tid] +=
                  cache_grad_attn_weight[tid + (s << 1)];
              cache_grad_sampling_loc[xid1] +=
                  cache_grad_sampling_loc[xid2 + (s << 1)];
              cache_grad_sampling_loc[xid1 + 1] +=
                  cache_grad_sampling_loc[xid2 + 1 + (s << 1)];
            }
          }
          __syncthreads();
        }

        if (tid == 0) {
          atomicAdd(grad_sampling_loc, cache_grad_sampling_loc[0]);
          atomicAdd(grad_sampling_loc + 1, cache_grad_sampling_loc[1]);
          atomicAdd(grad_attn_weight, cache_grad_attn_weight[0]);
        }
        __syncthreads();

        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
        grad_attn_weight += grad_weight_stride;
        grad_sampling_loc += grad_loc_stride;
      }
    }
  }
}

template <typename scalar_t>
__global__ void deform_col2im_gpu_kernel_gm(
    const int64_t n, const scalar_t *grad_col, const scalar_t *data_value,
    const int64_t *data_spatial_shapes, const int64_t *data_level_start_index,
    const scalar_t *data_sampling_loc, const scalar_t *data_attn_weight,
    const int64_t batch_size, const int64_t spatial_size,
    const int64_t num_heads, const int64_t channels, const int64_t num_levels,
    const int64_t num_query, const int64_t num_point, scalar_t *grad_value,
    scalar_t *grad_sampling_loc, scalar_t *grad_attn_weight) {
  CUDA_KERNEL_LOOP(index, n) {
    int64_t _temp = index;
    const int64_t c_col = _temp % channels;
    _temp /= channels;
    const int64_t sampling_index = _temp;
    const int64_t m_col = _temp % num_heads;
    _temp /= num_heads;
    // const int64_t q_col = _temp % num_query;
    _temp /= num_query;
    const int64_t b_col = _temp;

    const scalar_t top_grad = grad_col[index];

    int64_t data_weight_ptr = sampling_index * num_levels * num_point;
    int64_t data_loc_w_ptr = data_weight_ptr << 1;
    const int64_t grad_sampling_ptr = data_weight_ptr;
    grad_sampling_loc += grad_sampling_ptr << 1;
    grad_attn_weight += grad_sampling_ptr;
    const int64_t grad_weight_stride = 1;
    const int64_t grad_loc_stride = 2;
    const int64_t qid_stride = num_heads * channels;
    const int64_t data_value_ptr_init_offset =
        b_col * spatial_size * qid_stride;

    for (int l_col = 0; l_col < num_levels; ++l_col) {
      const int64_t level_start_id = data_level_start_index[l_col];
      const int64_t spatial_h_ptr = l_col << 1;
      const int64_t spatial_h = data_spatial_shapes[spatial_h_ptr];
      const int64_t spatial_w = data_spatial_shapes[spatial_h_ptr + 1];
      const int64_t value_ptr_offset =
          data_value_ptr_init_offset + level_start_id * qid_stride;
      const scalar_t *data_value_ptr = data_value + value_ptr_offset;
      scalar_t *grad_value_ptr = grad_value + value_ptr_offset;

      for (int p_col = 0; p_col < num_point; ++p_col) {
        const scalar_t loc_w = data_sampling_loc[data_loc_w_ptr];
        const scalar_t loc_h = data_sampling_loc[data_loc_w_ptr + 1];
        const scalar_t weight = data_attn_weight[data_weight_ptr];

        const scalar_t h_im = loc_h * spatial_h - 0.5;
        const scalar_t w_im = loc_w * spatial_w - 0.5;
        if (h_im > -1 && w_im > -1 && h_im < spatial_h && w_im < spatial_w) {
          col2im_bilinear_gm(data_value_ptr, spatial_h, spatial_w, num_heads,
                             channels, h_im, w_im, m_col, c_col, top_grad,
                             weight, grad_value_ptr, grad_sampling_loc,
                             grad_attn_weight);
        }
        data_weight_ptr += 1;
        data_loc_w_ptr += 2;
        grad_attn_weight += grad_weight_stride;
        grad_sampling_loc += grad_loc_stride;
      }
    }
  }
}

template <typename scalar_t>
void deform_im2col_cuda(hipStream_t stream, 
                        const scalar_t *data_value, // (B, N, G, D)
                        const int64_t *data_spatial_shapes, // (L, 2)
                        const int64_t *data_level_start_index, // (L,)
                        const scalar_t *data_sampling_loc, // (N, L, P, 2)
                        const scalar_t *data_attn_weight, // (N, L, P)
                        const int64_t batch_size, const int64_t spatial_size,
                        const int64_t num_heads, const int64_t channels,
                        const int64_t num_levels, const int64_t num_query,
                        const int64_t num_point, scalar_t *data_col) {
  const int64_t num_kernels = batch_size * num_query * num_heads * channels;
  const int64_t num_actual_kernels =
      batch_size * num_query * num_heads * channels;
  const int64_t num_threads = CUDA_NUM_THREADS;
  deform_im2col_gpu_kernel<scalar_t>
      <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0, stream>>>(
          num_kernels, data_value, data_spatial_shapes, data_level_start_index,
          data_sampling_loc, data_attn_weight, batch_size, spatial_size,
          num_heads, channels, num_levels, num_query, num_point, data_col);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in deform_im2col_cuda: %s\n", hipGetErrorString(err));
  }
}

template <typename scalar_t>
void deform_col2im_cuda(
    hipStream_t stream, const scalar_t *grad_col, const scalar_t *data_value,
    const int64_t *data_spatial_shapes, const int64_t *data_level_start_index,
    const scalar_t *data_sampling_loc, const scalar_t *data_attn_weight,
    const int64_t batch_size, const int64_t spatial_size,
    const int64_t num_heads, const int64_t channels, const int64_t num_levels,
    const int64_t num_query, const int64_t num_point, scalar_t *grad_value,
    scalar_t *grad_sampling_loc, scalar_t *grad_attn_weight) {
  const int64_t num_threads =
      (channels > CUDA_NUM_THREADS) ? CUDA_NUM_THREADS : channels;
  const int64_t num_kernels = batch_size * num_query * num_heads * channels;
  const int64_t num_actual_kernels =
      batch_size * num_query * num_heads * channels;
  if (channels > 1024) {
    if ((channels & 1023) == 0) {
      deform_col2im_gpu_kernel_shm_reduce_v2_multi_blocks<scalar_t>
          <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads,
             num_threads * 3 * sizeof(scalar_t), stream>>>(
              num_kernels, grad_col, data_value, data_spatial_shapes,
              data_level_start_index, data_sampling_loc, data_attn_weight,
              batch_size, spatial_size, num_heads, channels, num_levels,
              num_query, num_point, grad_value, grad_sampling_loc,
              grad_attn_weight);
    } else {
      deform_col2im_gpu_kernel_gm<scalar_t>
          <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
             stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                       data_level_start_index, data_sampling_loc,
                       data_attn_weight, batch_size, spatial_size, num_heads,
                       channels, num_levels, num_query, num_point, grad_value,
                       grad_sampling_loc, grad_attn_weight);
    }
  } else {
    switch (channels) {
      case 1:
        deform_col2im_gpu_kernel_shm_blocksize_aware_reduce_v1<scalar_t, 1>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 2:
        deform_col2im_gpu_kernel_shm_blocksize_aware_reduce_v1<scalar_t, 2>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 4:
        deform_col2im_gpu_kernel_shm_blocksize_aware_reduce_v1<scalar_t, 4>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 8:
        deform_col2im_gpu_kernel_shm_blocksize_aware_reduce_v1<scalar_t, 8>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 16:
        deform_col2im_gpu_kernel_shm_blocksize_aware_reduce_v1<scalar_t, 16>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 32:
        deform_col2im_gpu_kernel_shm_blocksize_aware_reduce_v1<scalar_t, 32>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 64:
        deform_col2im_gpu_kernel_shm_blocksize_aware_reduce_v2<scalar_t, 64>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 128:
        deform_col2im_gpu_kernel_shm_blocksize_aware_reduce_v2<scalar_t, 128>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 256:
        deform_col2im_gpu_kernel_shm_blocksize_aware_reduce_v2<scalar_t, 256>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 512:
        deform_col2im_gpu_kernel_shm_blocksize_aware_reduce_v2<scalar_t, 512>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      case 1024:
        deform_col2im_gpu_kernel_shm_blocksize_aware_reduce_v2<scalar_t, 1024>
            <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads, 0,
               stream>>>(num_kernels, grad_col, data_value, data_spatial_shapes,
                         data_level_start_index, data_sampling_loc,
                         data_attn_weight, batch_size, spatial_size, num_heads,
                         channels, num_levels, num_query, num_point, grad_value,
                         grad_sampling_loc, grad_attn_weight);
        break;
      default:
        if (channels < 64) {
          deform_col2im_gpu_kernel_shm_reduce_v1<scalar_t>
              <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads,
                 num_threads * 3 * sizeof(scalar_t), stream>>>(
                  num_kernels, grad_col, data_value, data_spatial_shapes,
                  data_level_start_index, data_sampling_loc, data_attn_weight,
                  batch_size, spatial_size, num_heads, channels, num_levels,
                  num_query, num_point, grad_value, grad_sampling_loc,
                  grad_attn_weight);
        } else {
          deform_col2im_gpu_kernel_shm_reduce_v2<scalar_t>
              <<<GET_BLOCKS(num_actual_kernels, num_threads), num_threads,
                 num_threads * 3 * sizeof(scalar_t), stream>>>(
                  num_kernels, grad_col, data_value, data_spatial_shapes,
                  data_level_start_index, data_sampling_loc, data_attn_weight,
                  batch_size, spatial_size, num_heads, channels, num_levels,
                  num_query, num_point, grad_value, grad_sampling_loc,
                  grad_attn_weight);
        }
    }
  }
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in deform_col2im_cuda: %s\n", hipGetErrorString(err));
  }
}

at::Tensor forward_cuda(const at::Tensor &value,
                        const at::Tensor &spatial_shapes,
                        const at::Tensor &level_start_index,
                        const at::Tensor &sampling_loc,
                        const at::Tensor &attn_weight,
                        const int64_t im2col_step) {
  CHECK_INPUT(value)
  CHECK_INPUT(spatial_shapes)
  CHECK_INPUT(level_start_index)
  CHECK_INPUT(sampling_loc)
  CHECK_INPUT(attn_weight)

  // value (B, H*W, H, C)
  const int64_t batch = value.size(0);
  const int64_t spatial_size = value.size(1);
  const int64_t num_heads = value.size(2);
  const int64_t channels = value.size(3);

  // spatial_shapes (L, Q, ..., P)
  const int64_t num_levels = spatial_shapes.size(0);
  const int64_t num_query = sampling_loc.size(1);
  const int64_t num_point = sampling_loc.size(4);

  const int64_t im2col_step_ = std::min(batch, im2col_step);

  CHECK_DIVISIBLE(batch, im2col_step_);

  auto output =
      at::zeros({batch, num_query, num_heads, channels}, value.options());

  const int64_t batch_n = im2col_step_;
  auto output_n = output.view(
      {batch / im2col_step_, batch_n, num_query, num_heads, channels});

  auto per_value_size = spatial_size * num_heads * channels;
  auto per_sample_loc_size = num_query * num_heads * num_levels * num_point * 2;
  auto per_attn_weight_size = num_query * num_heads * num_levels * num_point;

  for (int n = 0; n < batch / im2col_step_; ++n) {
    auto columns = output_n.select(0, n);
    AT_DISPATCH_FLOATING_TYPES(
        value.scalar_type(), "deform2d_multiscale_forward_cuda", ([&] {
          deform_im2col_cuda(
              at::cuda::getCurrentCUDAStream(),
              value.data_ptr<scalar_t>() + n * im2col_step_ * per_value_size,
              spatial_shapes.data_ptr<int64_t>(),
              level_start_index.data_ptr<int64_t>(),
              sampling_loc.data_ptr<scalar_t>() +
                  n * im2col_step_ * per_sample_loc_size,
              attn_weight.data_ptr<scalar_t>() +
                  n * im2col_step_ * per_attn_weight_size,
              batch_n, spatial_size, num_heads, channels, num_levels, num_query,
              num_point, columns.data_ptr<scalar_t>());
        }));
  }

  output = output.view({batch, num_query, num_heads * channels});
  return output;
}

std::vector<at::Tensor> backward_cuda(const at::Tensor &value,
                                      const at::Tensor &spatial_shapes,
                                      const at::Tensor &level_start_index,
                                      const at::Tensor &sampling_loc,
                                      const at::Tensor &attn_weight,
                                      const at::Tensor &grad_output,
                                      const int64_t im2col_step) {
  CHECK_INPUT(value)
  CHECK_INPUT(spatial_shapes)
  CHECK_INPUT(level_start_index)
  CHECK_INPUT(sampling_loc)
  CHECK_INPUT(attn_weight)
  CHECK_INPUT(grad_output)

  const int64_t batch = value.size(0);
  const int64_t spatial_size = value.size(1);
  const int64_t num_heads = value.size(2);
  const int64_t channels = value.size(3);

  const int64_t num_levels = spatial_shapes.size(0);
  const int64_t num_query = sampling_loc.size(1);
  const int64_t num_point = sampling_loc.size(4);

  const int64_t im2col_step_ = std::min(batch, im2col_step);

  CHECK_DIVISIBLE(batch, im2col_step_);

  auto grad_value = at::zeros_like(value);
  auto grad_sampling_loc = at::zeros_like(sampling_loc);
  auto grad_attn_weight = at::zeros_like(attn_weight);

  const int64_t batch_n = im2col_step_;

  auto per_value_size = spatial_size * num_heads * channels;
  auto per_sample_loc_size = num_query * num_heads * num_levels * num_point * 2;
  auto per_attn_weight_size = num_query * num_heads * num_levels * num_point;
  auto grad_output_n = grad_output.view(
      {batch / im2col_step_, batch_n, num_query, num_heads, channels});

  for (int n = 0; n < batch / im2col_step_; ++n) {
    auto grad_output_g = grad_output_n.select(0, n);
    AT_DISPATCH_FLOATING_TYPES(
        value.scalar_type(), "deform2d_multiscale_backward_cuda", ([&] {
          deform_col2im_cuda(
              at::cuda::getCurrentCUDAStream(),
              grad_output_g.data_ptr<scalar_t>(),
              value.data_ptr<scalar_t>() + n * im2col_step_ * per_value_size,
              spatial_shapes.data_ptr<int64_t>(),
              level_start_index.data_ptr<int64_t>(),
              sampling_loc.data_ptr<scalar_t>() +
                  n * im2col_step_ * per_sample_loc_size,
              attn_weight.data_ptr<scalar_t>() +
                  n * im2col_step_ * per_attn_weight_size,
              batch_n, spatial_size, num_heads, channels, num_levels, num_query,
              num_point,
              grad_value.data_ptr<scalar_t>() +
                  n * im2col_step_ * per_value_size,
              grad_sampling_loc.data_ptr<scalar_t>() +
                  n * im2col_step_ * per_sample_loc_size,
              grad_attn_weight.data_ptr<scalar_t>() +
                  n * im2col_step_ * per_attn_weight_size);
        }));
  }

  return {grad_value, grad_sampling_loc, grad_attn_weight};
}
}  // namespace deform2d_multiscale

// Register CUDA implementation with the PyTorch custom operation dispatcher
TORCH_LIBRARY_IMPL(deformops, CUDA, m) {
  m.impl("deform2d_multiscale_forward", &deform2d_multiscale::forward_cuda);
  m.impl("deform2d_multiscale_backward", &deform2d_multiscale::backward_cuda);
}
